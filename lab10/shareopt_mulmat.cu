#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h> 

#define BLOCK_SIZE 16
#define TOLERANCE 1e-5

// 核函数声明
__global__ void share_mulmat_block(float* out, float* A, float* B, int n);
__global__ void share_mulmat_col(float* out, float* A, float* B, int n);
__global__ void share_mulmat_row(float* out, float* A, float* B, int n);
__global__ void ref_matmul(float* out, float* A, float* B, int n);

// 矩阵初始化与验证函数
void initializeMatrix(float* matrix, int n);
bool verifyResults(float* ref, float* test, int n);

double getCurrentTime() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000.0 + tv.tv_usec / 1000.0;
}

int main(int argc, char** argv) {
    if (argc != 2) {
        printf("Usage: %s <matrix_size>\n", argv[0]);
        return -1;
    }
    
    int n = atoi(argv[1]);  // 获取矩阵维度
    size_t size = n * n * sizeof(float);
    
    // 创建主机内存
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C_block = (float*)malloc(size);
    float *h_C_row = (float*)malloc(size);
    float *h_C_col = (float*)malloc(size);
    float *h_ref = (float*)malloc(size);  // 参考结果
    
    // 初始化输入矩阵
    initializeMatrix(h_A, n);
    initializeMatrix(h_B, n);
    
    // 在主机上计算参考结果
    double host_start = getCurrentTime();
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            float sum = 0.0f;
            for (int k = 0; k < n; k++) {
                sum += h_A[i * n + k] * h_B[k * n + j];
            }
            h_ref[i * n + j] = sum;
        }
    }
    double host_end = getCurrentTime();
    printf("Host matrix multiplication time: %.3f ms\n", host_end - host_start);
    
    // 创建设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    // 复制数据到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // 1. 测试分块矩阵乘法
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    double block_start = getCurrentTime();
    share_mulmat_block<<<gridDim, blockDim>>>(d_C, d_A, d_B, n);
    hipDeviceSynchronize();
    double block_end = getCurrentTime();
    printf("Block matrix multiplication time: %.3f ms\n", block_end - block_start);
    hipMemcpy(h_C_block, d_C, size, hipMemcpyDeviceToHost);
    verifyResults(h_ref, h_C_block, n) ? printf("Block result: CORRECT\n") : printf("Block result: INCORRECT\n");
    
    // 2. 测试按行矩阵乘法
    dim3 rowBlock(256);  // 每个块256个线程
    dim3 rowGrid((n + rowBlock.x - 1) / rowBlock.x);
    
    double row_start = getCurrentTime();
    share_mulmat_row<<<rowGrid, rowBlock, n*sizeof(float)>>>(d_C, d_A, d_B, n);
    hipDeviceSynchronize();
    double row_end = getCurrentTime();
    printf("Row matrix multiplication time: %.3f ms\n", row_end - row_start);
    hipMemcpy(h_C_row, d_C, size, hipMemcpyDeviceToHost);
    verifyResults(h_ref, h_C_row, n) ? printf("Row result: CORRECT\n") : printf("Row result: INCORRECT\n");
    
    // 3. 测试按列矩阵乘法
    dim3 colBlock(256);  // 每个块256个线程
    dim3 colGrid((n + colBlock.x - 1) / colBlock.x);
    
    double col_start = getCurrentTime();
    share_mulmat_col<<<colGrid, colBlock, n*sizeof(float)>>>(d_C, d_A, d_B, n);
    hipDeviceSynchronize();
    double col_end = getCurrentTime();
    printf("Col matrix multiplication time: %.3f ms\n", col_end - col_start);
    hipMemcpy(h_C_col, d_C, size, hipMemcpyDeviceToHost);
    verifyResults(h_ref, h_C_col, n) ? printf("Col result: CORRECT\n") : printf("Col result: INCORRECT\n");
    
    // 清理内存
    free(h_A); free(h_B); free(h_ref);
    free(h_C_block); free(h_C_row); free(h_C_col);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    
    return 0;
}

// 验证函数实现
bool verifyResults(float* ref, float* test, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (fabs(ref[i * n + j] - test[i * n + j]) > TOLERANCE) {
                printf("Mismatch at (%d, %d): ref=%.6f, test=%.6f\n", 
                      i, j, ref[i * n + j], test[i * n + j]);
                return false;
            }
        }
    }
    return true;
}

// 初始化矩阵实现
void initializeMatrix(float* matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            matrix[i * n + j] = static_cast<float>(rand()) / RAND_MAX;
        }
    }
}

__global__ void share_mulmat_block(float* out, float* A, float* B, int n) {
    // 静态声明共享内存（大小在编译时确定）
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    // 计算当前线程要处理的全局位置
    int row = by * BLOCK_SIZE + ty;
    int col = bx * BLOCK_SIZE + tx;
    
    float dot = 0.0f;
    
    // 遍历所有分块
    for (int k = 0; k < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; k++) {
        // 加载A的分块（当前线程加载一个元素）
        int A_col = k * BLOCK_SIZE + tx;
        if (row < n && A_col < n) {
            As[ty][tx] = A[row * n + A_col];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        // 加载B的分块（当前线程加载一个元素）
        int B_row = k * BLOCK_SIZE + ty;
        if (B_row < n && col < n) {
            Bs[ty][tx] = B[B_row * n + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads(); // 确保所有线程完成加载
        
        // 计算当前分块的贡献（点积）
        for (int i = 0; i < BLOCK_SIZE; i++) {
            dot += As[ty][i] * Bs[i][tx];
        }
        
        __syncthreads(); // 确保所有线程完成计算
    }
    
    // 将结果写回全局内存
    if (row < n && col < n) {
        out[row * n + col] = dot;
    }
}

__global__ void share_mulmat_row(float* out, float* A, float* B, int n) {
    extern __shared__ float B_col[];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n) return;

    for (int col = 0; col < n; col++) {
        // 协作加载B的一列到共享内存
        for (int ii = 0; ii < n; ii += blockDim.x) {
            int idx = threadIdx.x + ii;
            if (idx < n) {
                B_col[idx] = B[idx * n + col];  // 加载B的第col列
            }
        }
        __syncthreads();  // 确保所有线程完成加载

        // 计算点积
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B_col[k];
        }
        out[row * n + col] = sum;

        __syncthreads();  // 确保所有线程完成计算
    }
}

__global__ void share_mulmat_col(float* out, float* A, float* B, int n) {
    extern __shared__ float A_row[];

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= n) return;  // 边界检查

    for (int row = 0; row < n; row++) {
        // 协作加载A的一行到共享内存
        for (int ii = 0; ii < n; ii += blockDim.x) {
            int idx = threadIdx.x + ii;
            if (idx < n) {
                A_row[idx] = A[row * n + idx];  // 合并内存访问
            }
        }
        __syncthreads();  // 确保所有线程完成加载

        // 计算点积
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += A_row[k] * B[k * n + col];  // 注意B的访问模式
        }
        out[row * n + col] = sum;

        __syncthreads();  // 确保所有线程完成计算
    }
}