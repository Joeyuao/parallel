#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h> 
#define BDIM 32
__global__ void trans(int* out, int* in, int n) {
    int bx = blockDim.x * blockIdx.x;
    int by = blockDim.y * blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by + ty;
    int col = bx + tx;
    
    if (row < n && col < n) {
        out[col * n + row] = in[row * n + col];
    }
}

__global__ void trans_conflict(int* out, int* in, int n) {
    __shared__ int smem[BDIM * BDIM];

    int bx = blockDim.x * blockIdx.x;
    int by = blockDim.y * blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by + ty;
    int col = bx + tx;
    
    if (row < n && col < n) {
        smem[ty*BDIM + tx] = in[row * n + col];
    }
    __syncthreads();
    
    if (row < n && col < n) {
        out[(bx+ty)*n + by+tx] = smem[tx*BDIM + ty];
    }
}

__global__ void trans_solve_conflict0(int* out, int* in, int n) {
    __shared__ int smem[BDIM * BDIM];

    int bx = blockDim.x * blockIdx.x;
    int by = blockDim.y * blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by + ty;
    int col = bx + tx;
    
    if (row < n && col < n) {
        smem[ty*BDIM + tx] = in[row * n + col];
    }
    __syncthreads();
    
    if (row < n && col < n) {
        out[col * n + row] = smem[ty*BDIM + tx];
    }
}

__global__ void trans_solve_conflict1(int* out, int* in, int n) {
    __shared__ int smem[BDIM * (BDIM+1)];

    int bx = blockDim.x * blockIdx.x;
    int by = blockDim.y * blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by + ty;
    int col = bx + tx;
    
    if (row < n && col < n) {
        smem[ty*(BDIM+1) + tx] = in[row * n + col];
    }
    __syncthreads();
    
    if (row < n && col < n) {
        out[(bx+ty)*n + by+tx] = smem[tx*(BDIM+1) + ty];
    }
}
void initializeMatrix(int* matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            matrix[i*n + j] = i * n + j;  
        }
    }
}

void printMatrix(int* matrix, int n, int size) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%3d ", matrix[i*size + j]);
        }
        printf("\n");
    }
}
int frac(int n_2){
    double sq = sqrt(double(n_2));
    for (int i = sq; i >= 1; i--){
        if(n_2 % i == 0){
            return i;
        }
    }
    return -1;
}
double getCurrentTime() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000.0 + tv.tv_usec / 1000.0;
}
int main(int argc,char**argv) {
    if (argc != 3) {
        printf("Usage: %s <matrix_size> <block_size>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int B_size = atoi(argv[2]);
    size_t size = n * n * sizeof(int);
    
    // 记录程序总开始时间
    double total_start = getCurrentTime();
    
    // 分配和初始化主机内存
    int* h_in = (int*)malloc(size);
    int* h_out = (int*)malloc(size);
    initializeMatrix(h_in, n);
    
    // 分配设备内存
    int *d_in, *d_out;
    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);
    
    // 拷贝数据到设备
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    
    // 计算网格和块维度
    int G_size2 = (n + B_size*B_size-1) / (B_size*B_size);
    int G_size_x = frac(G_size2);
    int G_size_y = G_size2 / G_size_x;
    dim3 BlockDim(B_size, B_size);
    dim3 GridDim(G_size_x, G_size_y);
    
    // 创建CUDA事件用于精确计时
    hipEvent_t start, stop;
    float avg = 0.0;
    float cnt = 111.0;
    for (float i = 0; i <= cnt; i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        // 记录内核开始时间
        hipEventRecord(start);
        
        // 启动内核
        trans_solve_conflict1<<<GridDim, BlockDim>>>(d_out, d_in, n);
        
        // 记录内核结束时间
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        // 计算内核执行时间
        float kernel_time = 0;
        hipEventElapsedTime(&kernel_time, start, stop);
        avg += kernel_time;
    }
    avg = avg / cnt;    
    
    // 拷贝结果回主机
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    
    // 记录程序总结束时间
    double total_end = getCurrentTime();
    
    // 打印结果验证
    printf("Original matrix (top-left 5x5):\n");
    printMatrix(h_in, 5, n);
    printf("\nTransposed matrix (top-left 5x5):\n");
    printMatrix(h_out, 5, n);
    
    // 打印计时结果
    printf("\nPerformance Metrics:\n");
    printf("Matrix size: %d x %d\n", n, n);
    printf("Block size: %d x %d\n", B_size, B_size);
    printf("Grid size: %d x %d\n", G_size_x, G_size_y);
    printf("Kernel execution time: %.6f ms\n", avg);
    printf("Total program time: %.3f ms\n", total_end - total_start);
    
    // 清理
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipDeviceSynchronize();
    return 0;
}